// TODOS:
// 1. Consider the case where n is not divisible by num_gpus
// 2. Compare time for async and non-async
// 3. Incorporate streams within each GPU computation
// 4. Change the initial kernel to handle n x m matrix
// 5. Make it take a matrix of n x m 
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "matmul.cuh"
#include "utils.cuh"
#include <string>
#include <hipblas.h>

void transpose(float *output, const float *input, int rows, int columns, int start_col, int end_col) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Calculate the number of columns to extract
    int num_cols_to_extract = end_col - start_col;

    // Use hipblasSgeam to extract the specified range of columns
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rows, num_cols_to_extract, &alpha,
                input + start_col * rows, rows, &beta, nullptr, rows,
                output, rows);
    hipDeviceSynchronize();
}

int main(int argc, char** argv)
{
    printf("Distributed matmul with managed memory\n");
    if (argc != 3){
        printf("Usage: ./t <matrix size> <num_gpus>\n");
        return 0;    
    }

    int n = std::stoi(argv[1]);
    // int threads_per_block = stoi(argv[2]);
    int threads_per_block = 1024;
    int num_gpus = std::stoi(argv[2]);
    
    // check n is divisible by num_gpus
    if (! (n % num_gpus == 0) ){
        printf("For now, only supports n divisible by num_gpus");
        return 0;    
    }
    /////////////////// hardcode params for testing ///////////////////
    printf("Hardcoding params for testing\n");
    printf("n=%d, num_gpus=%d\n", n, num_gpus);
    num_gpus = 2;
    int nRowsA = n, nColsA = n, nColsB = n; // test square matrices for now
    int matrix_size = num_gpus * nRowsA * nColsA; // Total size of matrix
    int chunk_size = matrix_size / num_gpus; // Chunk going on each GPU

    // grid and block sizes
    dim3 threadsPerBlock(threads_per_block);
    int blocks_per_dim = (chunk_size + threadsPerBlock.x - 1) / threadsPerBlock.x;
    dim3 blocksPerGrid(blocks_per_dim);
    /////////////////// hardcode params for testing ///////////////////
    
    // Set up operands and result on device 0 
    float* defaultArrA;
    float* defaultArrB;
    float* defaultArrC;
    float* hostArrayD;
    // Use managed for async memcpy
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&defaultArrA, matrix_size  * sizeof(float))); 
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&defaultArrB, matrix_size  * sizeof(float))); 
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&defaultArrC, matrix_size  * sizeof(float))); 
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&hostArrayD, matrix_size  * sizeof(float))); 

    // randomly init and rescale the array on GPU. Make a separate dim for memory allocation
    dim3 threadsPerBlockAlloc(threads_per_block);
    int blocks_per_dim_alloc = (matrix_size + threadsPerBlockAlloc.x - 1) / threadsPerBlockAlloc.x;
    dim3 blocksPerGridAlloc(blocks_per_dim_alloc);
    GPU_fill_rand_int<<<blocksPerGridAlloc, threadsPerBlockAlloc>>>(defaultArrA, matrix_size, 1.0f, 2.0f);
    GPU_fill_rand_int<<<blocksPerGridAlloc, threadsPerBlockAlloc>>>(defaultArrB, matrix_size, 0.0f, 0.0f);
    kernel_err_check();
    hipDeviceSynchronize();
    printMatrix(defaultArrA, nRowsA, nColsA);
    transpose(defaultArrB, defaultArrA, nRowsA, nColsA, 0, 8);
    printf("Printing matrix\n");
    printMatrix(defaultArrB, nColsA, nRowsA);
}