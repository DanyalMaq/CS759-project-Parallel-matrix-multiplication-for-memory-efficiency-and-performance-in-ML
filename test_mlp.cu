#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include "include/matmul.cuh"
#include "include/utils.cuh"
#include<complex>
#include "include/network.cuh"
#include "cnpy.h"

int main()
{
    MatrixDims* layer_dims = new MatrixDims[3];

    // Simple test network
    // layer_dims[0] = MatrixDims(100, 100);
    // layer_dims[1] = MatrixDims(100, 100);
    // layer_dims[2] = MatrixDims(100, 100);

    // // For propr network
    layer_dims[0] = MatrixDims(784, 150);
    layer_dims[1] = MatrixDims(150, 200);
    layer_dims[2] = MatrixDims(200, 10);

    float** mat_weights = new float*[3];
    string load_path[] = {"./data/data/linear_0.weight.npy", "./data/data/linear_1.weight.npy", "./data/data/linear_2.weight.npy"};

    for (int i = 0; i < 3; i++){
        mat_weights[i] = new float[layer_dims[i].nRows * layer_dims[i].nCols];
        cnpy::NpyArray arr = cnpy::npy_load(load_path[i]);
        mat_weights[i] = arr.data<float>();
    }
    
    uint32_t num_devices = 2;
    uint32_t num_layers = 3;
    uint32_t batch_size = 32;
    MLP network(num_layers, layer_dims, num_devices, mat_weights);
    network.enable_tp(batch_size);
}