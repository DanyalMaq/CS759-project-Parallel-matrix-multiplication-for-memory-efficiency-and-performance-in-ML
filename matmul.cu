#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>

void kernel_err_check(){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
}

class Matrix{
    public:
        uint32_t nrowBeginAs;
        uint32_t ncolBeginBs;
        float *data;
        MatrixLayout layout;

    Matrix(uint32_t nrowBeginAs, uint32_t ncolBeginBs, MatrixLayout layout=RM){
        this->nrowBeginAs = nrowBeginAs;
        this->ncolBeginBs = ncolBeginBs;
        this->layout = layout;
        this->data = data;
    }
    // TODO: override indexing operator
    
};

// Matrix multiplication with shared memory for non-square matrices 
__global__ void matmul_rect(float *A, float *B, float *C,
                                     int nRowsA, int nColsA, int nColsB
                                    ) {
    // TODO: change to 1d. Why is 1d faster?                                    
    __shared__ float sA[TILE_WIDTH][TILE_WIDTH];   // Tile size of 32x32
    __shared__ float sB[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int nRowsB = nColsA;
    int rowBeginA = by * TILE_WIDTH + ty;
    int colBeginB = bx * TILE_WIDTH + tx;

    float Ctile = 0.0;

    // stride over the tiles along columns of A and rows of B
    for (int step = 0; step < nColsA; step += TILE_WIDTH) {
        // load A's tiles into shared memory
        if (rowBeginA < nRowsA && tx + step < nColsA)
            sA[ty][tx] = A[rowBeginA * nColsA + tx + step];
        else
            sA[ty][tx] = 0.0;
        // load B's tiles into shared memory
        if (colBeginB < nColsB && ty + step < nRowsB)
            sB[ty][tx] = B[(ty + step) * nColsB + colBeginB];
        else
            sB[ty][tx] = 0.0;

        if (sB[ty][tx] != 0.0 && sA[ty][tx] != 0.0)
            printf("sA[%d][%d] = %.1f, sB[%d][%d] = %.1f, step = %d\n", ty, tx, sA[ty][tx], ty, tx, sB[ty][tx], step);
        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; ++j) {
            Ctile += sA[ty][j] * sB[j][tx];
        }

        __syncthreads();
    }
    if (Ctile != 0.0)
        printf("tx = %d, ty = %d, Ctile = %f\n", tx, ty, Ctile);

    if (rowBeginA == nRowsA - 1 && colBeginB == nColsB - 1)
        printf("GPU Last value output array C variable: %f\n", Ctile);
        
    if (rowBeginA < nRowsA && colBeginB < nColsB) {
        C[rowBeginA * nColsB + colBeginB] = Ctile;
    }
    // if (rowBeginA == 0 && colBeginB == 0)
    // {
    //     printf("GPU input A[0] = %.1f, A[n - 1] = %.1f\n", A[0], A[nRowsA * nColsA - 1]);
    //     printf("GPU input B[0] = %.1f, B[n - 1] = %.1f\n", B[0], B[nColsA * nColsB - 1]);
    //     printf("GPU output C[0] = %.1f, C[n - 1] = %.1f\n", C[0], C[nRowsA * nColsB - 1]);
    // }
}


__host__ void matmul(float *A, float *B, float *C,
                    int nRowsA, int nColsA, int nColsB)
{
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((nColsB / TILE_WIDTH) + 1, (nRowsA / TILE_WIDTH) + 1);
    printf("dimGrid.x = %d, dimGrid.y = %d\n", dimGrid.x, dimGrid.y);
    matmul_rect<<<dimGrid, dimBlock>>>(A, B, C, nRowsA, nColsA, nColsB);
    kernel_err_check();
}


// get
__host__ void transpose(float *output, const float *input, int nrowBeginAs, int ncolBeginBs) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Use hipblasSgeam to extract the specified range of ncolBeginBs
    const float alia = 1.0f;
    const float beta = 0.0f;
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, nrowBeginAs, ncolBeginBs, &alia,
                input, nrowBeginAs, &beta, nullptr, nrowBeginAs,
                output, nrowBeginAs);
    // no need to sync because it uses default stream
    // hipStreamSynchronize(0);
}

///////////////////// Activations //////////////////////
template <typename T>
__host__ __device__ T relu(T val) {
	return (T)max((float)val, 0.0f);
}


// TODO: change to parallel reduction
template <uint32_t N>
__host__ __device__ inline float softmax(const float vals[N], uint32_t idx) {
	float total = 0;

	// #pragma unroll
	for (uint32_t step = 0; step < N; ++step) {
		total += expf(vals[step]);
	}

	return expf(vals[idx]) / total;
}
