#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

void kernel_err_check(){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
}


const unsigned int TILE_WIDTH = 32; // Tile size of shared memory

__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int nRowsA, int nColsA, int nColsB
                                     ) {
    __shared__ float sA[TILE_WIDTH][TILE_WIDTH];   // Tile size of 32x32
    __shared__ float sB[TILE_WIDTH][TILE_WIDTH];

    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int ph = 0; ph < (((nColsA - 1) / TILE_WIDTH) + 1); ph++) {
        if ((Row < nRowsA) && (threadIdx.x + (ph * TILE_WIDTH)) < nColsA) {
            sA[threadIdx.y][threadIdx.x] = A[(Row * nColsA) + threadIdx.x + (ph * TILE_WIDTH)];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (Col < nColsB && (threadIdx.y + ph * TILE_WIDTH) < nColsA) {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + ph * TILE_WIDTH) * nColsB + Col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; ++j) {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }

    if (Row == (nRowsA-1) && Col == (nColsB-1))
    {
        printf("GPU Last value output array C variable: %f\n", Cvalue);
    }
    if (Row < nRowsA && Col < nColsB) {
        C[Row * nColsB + Col] = Cvalue;
    }
    if (Row == 0 && Col == 0)
    {
        printf("GPU First value input array A: %f\n", A[0]);
        printf("GPU First value output array C: %f\n", C[0]);
    }
}


__host__ void matmul(float *A, float *B, float *C,
                                     int nRowsA, int nColsA, int nColsB)
{
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((nColsB / TILE_WIDTH) + 1, (nRowsA / TILE_WIDTH) + 1, 1);

    matrixMultiplyShared<<<dimGrid, dimBlock>>>(A, B, C, nRowsA, nColsA, nColsB);
    kernel_err_check();
    hipDeviceSynchronize();
}

// Fill an array with random integers in [min, max]
__global__ void  GPU_fill_rand_int(float* A, const int n, float min, float max) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    // Initialize the random state for the current thread
    hiprandState state;
    unsigned long long seed = 759;
    hiprand_init(seed, idx, 0, &state);
    
    // Generate a random float and convert it to an integer
    float rnd = hiprand_uniform(&state); // (0.0, 1.0]
    A[idx] = static_cast<int>( rnd * (max - min) + min );
}



////////////////// helper functions //////////////////////
__global__ void addOneToElements(int* array, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        array[index] += 1;
    }
}

__host__ void printMatrix(float* array, int n)
{
    printf("Matrix:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f ", array[i*n + j]);
        }
       printf("\n");
    }
}
