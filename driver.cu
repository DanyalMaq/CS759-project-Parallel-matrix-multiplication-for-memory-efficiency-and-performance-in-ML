#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <random>

int main(int argc, char* argv[]) {
    // int n = atoi(argv[1]);
    // int threads = atoi(argv[2]);
    float ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int n = 1000;
    int m = 200;

    // float
    float* Afloat = new float[n*m];
    float* Bfloat = new float[m*n];;
    float* Cfloat = new float[n*n];;
    float* AfloatD;
    float* BfloatD;
    float* CfloatD;
    // hipMallocManaged(&Afloat, (n * m) * sizeof(float));
    // hipMallocManaged(&Bfloat, (n * m) * sizeof(float));
    // hipMallocManaged(&Cfloat, (n * m) * sizeof(float));
    
    for (unsigned int i = 0; i < n * m; i++) {
        Afloat[i] = 1.0;
    }
    for (unsigned int i = 0; i < m * n; i++) {
        Bfloat[i] = 1.0;
    }
    for (unsigned int i = 0; i < n * n; i++) {
        Cfloat[i] = 0.0;
    }

    hipMalloc((void**)&AfloatD, n * m * sizeof(float));
    hipMemcpy(AfloatD, Afloat, n * m * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&BfloatD, m * n * sizeof(float));
    hipMemcpy(BfloatD, Bfloat, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&CfloatD, n * n * sizeof(float));
    hipMemcpy(CfloatD, Cfloat, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start);
    matmul(AfloatD, BfloatD, CfloatD, n, m, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(Cfloat, CfloatD, n * n * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("%f\n%f\n%f\n", Cfloat[0], Cfloat[n * n - 1], ms);

    hipFree(AfloatD);
    hipFree(BfloatD);
    hipFree(CfloatD);
}