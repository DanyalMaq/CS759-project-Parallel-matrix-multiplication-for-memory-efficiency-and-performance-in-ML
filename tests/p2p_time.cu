#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include "../matmul.cuh"
#include "../utils.cuh"
#include "../network.cuh"
using namespace std;

__global__ void test_memory(volatile float* arr, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        arr[i] = 1.0;
    }
}

int main(int argc, char** argv){
    // check n is divisible by num_gpus
    /////////////////// hardcode params for testing ///////////////////
    int num_gpus = 2, n = 16384; // 2^14 * 2^14 matrix
    int th_per_block = 1024;
    int nRowsA = n, nColsA = n, nColsB = n; // test square matrices for now
    int matrix_size = num_gpus * nRowsA * nColsA; // Total size of matrix

    printf("Hardcoding params for testing\n");
    printf("n = %d, num_gpus = %d\n", n, num_gpus);

    // grid and block sizes
    dim3 threadsPerBlock(th_per_block);
    int blocks_per_dim = (matrix_size + threadsPerBlock.x - 1) / threadsPerBlock.x;
    dim3 blocksPerGrid(blocks_per_dim);
    // Set up operands and result on device 0 
    float* defaultArrA;
    float* deviceArr;
    float* hostArr = (float*)malloc(100 * sizeof(float));

    hipEvent_t start, stop;
    float time;

    // Use managed for async memcpy
    hipSetDevice(0);
    CHECK_CUDA_ERROR(hipMalloc((void**)&defaultArrA, matrix_size  * sizeof(float))); 
    
    // Enable bi-directional peer access
    set_p2p_access(num_gpus);
    
    // Test access speed from peer device without prefetching
    hipSetDevice(1);
    // Allocate memory on device 1
    CHECK_CUDA_ERROR(hipMalloc((void**)&deviceArr, matrix_size  * sizeof(float)));
    CHECK_CUDA_ERROR(hipEventCreate(&start)); CHECK_CUDA_ERROR(hipEventCreate(&stop));
    CHECK_CUDA_ERROR(hipEventRecord(start));

    test_memory<<<blocksPerGrid, threadsPerBlock>>>(defaultArrA, matrix_size);
    kernel_err_check();

    // End of timing
    hipEventRecord(stop);
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    printf("Time to access peer device memory without prefetching:  %3.1f ms \n", time);

    // Check device id    
    hipPointerAttribute_t attr;
    CHECK_CUDA_ERROR(hipPointerGetAttributes(&attr, defaultArrA));
    printf("Device id of arr before copy: %d\n", attr.device);
    CHECK_CUDA_ERROR(hipPointerGetAttributes(&attr, hostArr));
    printf("Device id of host array: %d\n", attr.device);

    // Time p2p memcpy
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMemcpyPeer(defaultArrA, 0, deviceArr, 1, matrix_size * sizeof(float)));
    hipEventRecord(stop);
    
    // Check device id
    hipPointerGetAttributes(&attr, deviceArr);
    printf("Device id of arr after copy: %d\n", attr.device);
    // End of timing
    hipEventSynchronize(stop);
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    printf("Time to hipMemcpyPeer %lu bytes:  %3.1f ms \n", matrix_size * sizeof(float), time);
    
    // with prefetching
    hipEventRecord(start);
    test_memory<<<blocksPerGrid, threadsPerBlock>>>(deviceArr, matrix_size);
    kernel_err_check();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    printf("Time to access peer device memory with prefetching:  %3.1f ms \n", time);

    // Finally,test normal copy speed instead of p2p copy
    set_p2p_access(false);
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMemcpy(defaultArrA, deviceArr, matrix_size * sizeof(float), hipMemcpyDeviceToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time to hipMemcpy  %lu bytes:  %3.1f ms \n", matrix_size * sizeof(float), time);

}