#include "../src/matmul.cuh"
#include "../src/utils.cuh"
#include <iostream>

int main(){
    float* a;
    CHECK_CUDA_ERROR(hipSetDevice(0));
    CHECK_CUDA_ERROR(hipMalloc(&a, 4*4*sizeof(float)));
    CHECK_CUDA_ERROR(hipSetDevice(1));
    // test error
    CHECK_CUDA_ERROR(hipFree(a));
    cout<< a[1];
}