#include "hip/hip_runtime.h"
// TODOS:
// 1. Consider the case where n is not divisible by numGPUs
// 2. Compare time for async and non-async
// 3. Incorporate streams within each GPU computation
// 4. Change the initial kernel to handle n x m matrix
// 5. Make it take a matrix of n x m 
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "matmul.cuh"
#include <string>
using namespace std;

__global__ void addOneToElements(int* array, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        array[index] += 1;
    }
}

__host__ void printMatrix(float* array, int n)
{
    printf("Matrix:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f ", array[i*n + j]);
        }
        printf("\n");
    }
}


#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}


int main(int argc, char** argv){
    if (argc != 3){
        printf("Usage: ./t <matrix size> <numGPUs>\n");
        return 0;    
    }

    int n = std::stoi(argv[1]);
    // int threads_per_block = stoi(argv[2]);
    int threads_per_block = 1024;
    int numGPUs = stoi(argv[2]);
    
    // check n is divisible by numGPUs
    if (! (n % numGPUs == 0) ){
        printf("For now, only supports n divisible by numGPUs");
        return 0;    
    }
    /////////////////// hardcode params for testing ///////////////////
    printf("Hardcoding params for testing\n");
    numGPUs = 2;
    int nRowsA = n, nColsA = n, nColsB = n; // test square matrices for now
    int matrix_size = numGPUs * nRowsA * nColsA; // Total size of matrix
    int chunk_size = matrix_size + n - 1 / numGPUs; // Chunk going on each GPU

    // grid and block sizes
    dim3 threadsPerBlock(threads_per_block);
    int blocks_per_dim = (chunk_size + threadsPerBlock.x - 1) / threadsPerBlock.x;
    dim3 blocksPerGrid(blocks_per_dim);
    /////////////////// hardcode params for testing ///////////////////
    
    // Set up operands and result on device 0 
    float* hostArrayA;
    float* hostArrayB;
    float* hostArrayC;
    hipMallocManaged((void**)&hostArrayA, matrix_size  * sizeof(float)); 
    hipMallocManaged((void**)&hostArrayB, matrix_size  * sizeof(float)); 
    hipMallocManaged((void**)&hostArrayC, matrix_size  * sizeof(float)); 

    // randomly init and rescale the array on GPU
    GPU_fill_rand<<<blocksPerGrid, threadsPerBlock>>>(hostArrayA, matrix_size, 1.0f, 1.0f);
    GPU_fill_rand<<<blocksPerGrid, threadsPerBlock>>>(hostArrayB, matrix_size, 1.0f, 1.0f);
    hipDeviceSynchronize();
    printf("First value input: %f\nLast value input: %f\n", hostArrayA[0], hostArrayA[matrix_size-1]);
    
    hipStream_t streams[numGPUs]; // Create a stream for each GPU for overlapping
    float* deviceArraysA[numGPUs];
    float* deviceArraysB[numGPUs];
    float* deviceArraysC[numGPUs];

    // Allocate chunk on each GPU
    for (int i = 1; i < numGPUs; ++i) {
        hipSetDevice(i);
        hipMallocAsync((void**)&deviceArraysA[i], chunk_size * sizeof(float), 0);
        hipMallocAsync((void**)&deviceArraysB[i], chunk_size * sizeof(float), 0);
        hipMallocAsync((void**)&deviceArraysC[i], chunk_size * sizeof(float), 0);
    }


    // enable access from device 0 to all others
    hipSetDevice(0);
    for (int i = 1; i < numGPUs; ++i) {
        int start = i * chunk_size;
        if (i != 0){ // funny but...self-to-self access will fail
            int canAccess = 0;
            CHECK_CUDA_ERROR(hipDeviceEnablePeerAccess(i, 0));
            hipMemcpyPeerAsync(deviceArraysA[i], i, (hostArrayA + start), 0, chunk_size * sizeof(float), 0);
            hipMemcpyPeerAsync(deviceArraysB[i], i, (hostArrayB + start), 0, chunk_size * sizeof(float), 0);
        }
    }

    // Launch kernel on each GPU with appropriate configurations
    for (int i = 0; i < numGPUs; ++i) {  
        hipSetDevice(i);
        int start = i * chunk_size;
        int end = start + chunk_size;
    
        if (i == 0)
        {
            matmul(hostArrayA, hostArrayB, hostArrayC, nRowsA, nColsA, nColsB);
        }
        else
        {
            // call matmul on device i for the chunk
            // unsigned int shared_mem_size = 2 * sizeof(float) * (blocks_per_dim / numGPUs) * (blocks_per_dim / numGPUs);
            matmul(deviceArraysA[i], deviceArraysB[i], deviceArraysC[i], nRowsA, nColsA, nColsB);
            hipSetDevice(0); // ensure correct copying to default device
            hipMemcpyPeerAsync(hostArrayC + start, 0, deviceArraysC[i], i, chunk_size * sizeof(float), 0);
        }
    }
 
    // wait for results
    for (int i = 0; i < numGPUs; ++i) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }
    
    //Print the result
    // printMatrix(hostArrayC, n);
    printf("First value output: %f\nLast value output: %f\n", hostArrayC[0], hostArrayC[matrix_size-1]);
    
    // Free allocated memory
    delete[] hostArrayA;
    delete[] hostArrayB;
    delete[] hostArrayC;
    for (int i = 0; i < numGPUs; ++i) {
        hipSetDevice(i);
        hipFree(deviceArraysA[i]);
        hipFree(deviceArraysB[i]);
        hipFree(deviceArraysC[i]);
    }

    return 0;
}