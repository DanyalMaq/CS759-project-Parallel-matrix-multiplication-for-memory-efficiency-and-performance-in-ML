#include "hip/hip_runtime.h"
// TODOS:
// 1. Consider the case where n is not divisible by numGPUs
// 2. Compare time for async and non-async
// 3. Incorporate streams within each GPU computation
// 4. Change the initial kernel to handle n x m matrix
// 5. Make it take a matrix of n x m 
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "matmul.cuh"
#include <string>
using namespace std;

__global__ void addOneToElements(int* array, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        array[index] += 1;
    }
}

__host__ void printMatrix(float* array, int n)
{
    printf("Matrix:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f ", array[i*n + j]);
        }
        printf("\n");
    }
}


#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}


int main(int argc, char** argv){
    if (argc != 3){
        printf("Usage: ./t <matrix size> <numGPUs>\n");
        return 0;    
    }

    int n = std::stoi(argv[1]);
    // int threads_per_block = stoi(argv[2]);
    int threads_per_block = 1024;
    int numGPUs = stoi(argv[2]);
    
    // check n is divisible by numGPUs
    if (! (n % numGPUs == 0) ){
        printf("For now, only supports n divisible by numGPUs");
        return 0;    
    }
    /////////////////// hardcode params for testing ///////////////////
    printf("Hardcoding params for testing\n");
    numGPUs = 2;
    int nRowsA = n, nColsA = n, nColsB = n;
    int matrix_size = numGPUs * nRowsA * nColsA; // test square matrices for now
    int chunk_size = (matrix_size / numGPUs);

    // grid and block sizes
    dim3 threadsPerBlock(threads_per_block);
    int blocks_per_dim = (chunk_size + threadsPerBlock.x - 1) / threadsPerBlock.x;
    dim3 blocksPerGrid(blocks_per_dim);
    /////////////////// hardcode params for testing ///////////////////
    
    // Set up operands and result on device 0 
    float* hostArrayA;
    float* hostArrayB;
    float* hostArrayC;
    hipMallocManaged((void**)&hostArrayA, matrix_size  * sizeof(float)); 
    hipMallocManaged((void**)&hostArrayB, matrix_size  * sizeof(float)); 
    hipMallocManaged((void**)&hostArrayC, matrix_size  * sizeof(float)); 

    // randomly init and rescale the array on GPU
    GPU_fill_rand_int<<<blocksPerGrid, threadsPerBlock>>>(hostArrayA, matrix_size, 1.0f, 1.0f);
    GPU_fill_rand_int<<<blocksPerGrid, threadsPerBlock>>>(hostArrayB, matrix_size, 1.0f, 1.0f);
    
    hipStream_t streams[numGPUs]; // Create a stream for each GPU for overlapping
    float* deviceArraysA[numGPUs];
    float* deviceArraysB[numGPUs];
    float* deviceArraysC[numGPUs];

    // Allocate chunk on each GPU
    for (int i = 0; i < numGPUs; ++i) {
        hipSetDevice(i);
        hipMallocAsync((void**)&deviceArraysA[i], (matrix_size + n - 1) / numGPUs  * sizeof(float), 0);
        hipMallocAsync((void**)&deviceArraysB[i], (matrix_size + n - 1) / numGPUs  * sizeof(float), 0);
        hipMallocAsync((void**)&deviceArraysC[i], (matrix_size + n - 1) / numGPUs  * sizeof(float), 0);
    }


    // enable access from device 0 to all others
    hipSetDevice(0);
    for (int i = 0; i < numGPUs; ++i) {
        int start = i * chunk_size;
        if (i != 0){ // funny but...self-to-self access will fail
            int canAccess = 0;
            CHECK_CUDA_ERROR(hipDeviceEnablePeerAccess(i, 0));
            hipMemcpyPeerAsync(deviceArraysA[i], i, (hostArrayA + start), 0, chunk_size * sizeof(float), 0);
            hipMemcpyPeerAsync(deviceArraysB[i], i, (hostArrayB + start), 0, chunk_size * sizeof(float), 0);
        }
    }

    // Launch kernel on each GPU with appropriate configurations
    for (int i = 0; i < numGPUs; ++i) {  
        hipSetDevice(i);
        int start = i * chunk_size;
        int end = start + chunk_size;
    
        // call matmul on device i for the chunk
        // unsigned int shared_mem_size = 2 * sizeof(float) * (blocks_per_dim / numGPUs) * (blocks_per_dim / numGPUs);
        matmul(deviceArraysA[i], deviceArraysB[i], deviceArraysC[i], nRowsA, nColsA, nColsB);
        
        hipSetDevice(0); // ensure correct copying to default device
        hipMemcpyPeerAsync(hostArrayC + start, 0, deviceArraysC[i], i, chunk_size * sizeof(float), 0);
    }
 
    // wait for results
    for (int i = 0; i < numGPUs; ++i) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }
    
    //Print the result
    printMatrix(hostArrayC, n);
    // printf("%f\n%f", hostArrayC[0], hostArrayC[n]);
    
    // Free allocated memory
    delete[] hostArrayA;
    delete[] hostArrayB;
    delete[] hostArrayC;
    for (int i = 0; i < numGPUs; ++i) {
        hipSetDevice(i);
        hipFree(deviceArraysA[i]);
        hipFree(deviceArraysB[i]);
        hipFree(deviceArraysC[i]);
    }

    return 0;
}